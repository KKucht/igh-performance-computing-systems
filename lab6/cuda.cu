#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

__global__ 
void checkIfPrime(unsigned long int* inputArgument, unsigned long int* returnArg){
  unsigned long int index=blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long int my_number = (unsigned long int)inputArgument[index];
  if (my_number == 1 || my_number == 0) {
    returnArg[index] = 0;
    return;
  }

  if (my_number < 4){
    returnArg[index] = 1;
    return;
  }
    
  long range = my_number / 2 + 1;
  for (long i = 2 ; i*i <= my_number;  i++){
    if (my_number % i == 0 ){
      returnArg[index] = 0;
      return;
    }
  }
  returnArg[index] = 1;
  return;
}



int main(int argc,char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);

  int threadsinblock=1024;
  int blocksingrid=10000;	
  long long result;
  long long size = threadsinblock*blocksingrid;
  
  //program input argument
  long inputArgument = ins__args.arg; 
  unsigned long int *numbers = (unsigned long int*)malloc(size * sizeof(unsigned long int));
  for (int i =0;i<size;i++)
    numbers[i]=0;

  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);
  
  // run your CUDA kernel(s) here

    
    
    //memory allocation on host
    unsigned long int *hresults=(unsigned long int*)malloc(size*sizeof(unsigned long int));
    if (!hresults) errorexit("Error allocating memory on the host");	

    unsigned long int *dresults=NULL;
    unsigned long int *dnumbers=NULL;

    if (hipSuccess!=hipMalloc((void **)&dresults,size*sizeof(unsigned long int)))
      errorexit("Error allocating memory on the GPU");

    if (hipSuccess!=hipMalloc((void **)&dnumbers,size*sizeof(unsigned long int)))
      errorexit("Error allocating memory on the GPU");

    if (hipSuccess!=hipMemcpy(dnumbers,numbers,size*sizeof(unsigned long int),hipMemcpyHostToDevice))
       errorexit("Error copying numbers");

    checkIfPrime<<<blocksingrid,threadsinblock>>>(dnumbers, dresults);
    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");
  
    if (hipSuccess!=hipMemcpy(hresults,dresults,size*sizeof(unsigned long int),hipMemcpyDeviceToHost))
       errorexit("Error copying results ");


    //calculate sum of all elements on CPU side
    result=0;

    for(int i=0;i<size;i++) {
      result += hresults[i];
    }

    printf("\nThe final result is %lld\n",result);

    // synchronize/finalize your CUDA computations

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);


}
